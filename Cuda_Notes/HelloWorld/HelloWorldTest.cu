#include "hip/hip_runtime.h"
# Date : 2023/04/20 
# Author : Fangjun Wen 
# Function : Test Cuda Toolkit 

#include <iostream>

__global__ void kernel( void ) {
	printf( "Hello, World!\n" );
}

int main( void ) {
	kernel<<<1,1>>>();

	return 0;
}