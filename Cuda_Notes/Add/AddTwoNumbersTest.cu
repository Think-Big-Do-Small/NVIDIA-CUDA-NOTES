#include "hip/hip_runtime.h"
# Date : 2023/04/20 
# Author : Fangjun Wen 
# Function : Cuda Add Two Numbers  

#include <iostream>


__global__ void add(int *a, int *b, int *c)
{
	*c = *a + *b; 
}


void get_add_result(int a, int b, int & c)
{
	int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
	int size = sizeof( int ); // we need space for an integer
	// allocate device copies of a, b, c
	hipMalloc( (void**)&dev_a, size );
	hipMalloc( (void**)&dev_b, size );
	hipMalloc( (void**)&dev_c, size );

	// copy inputs to device
	hipMemcpy( dev_a, &a, size, hipMemcpyHostToDevice );
	hipMemcpy( dev_b, &b, size, hipMemcpyHostToDevice );
	// launch add() kernel on GPU, passing parameters
	add<<< 1, 1 >>>( dev_a, dev_b, dev_c );
	hipDeviceSynchronize();
	
	// copy device result back to host copy of c
	hipMemcpy( &c, dev_c, size, hipMemcpyDeviceToHost );

	hipFree( dev_a ); hipFree( dev_b ); hipFree( dev_c );
}


int main( void ) {
	int a,b; int sum; 
	printf("Enter a and b:");
	scanf("%d,%d",&a,&b);
	
	get_add_result(a,b,sum); 
	printf("sum = %d\n", sum); 

	return 0;
}